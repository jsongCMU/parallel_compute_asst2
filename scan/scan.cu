#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

const int num_print = 128; // DEBUGGING
void print_host_data(int *data, int size, int num_print)
{
    // Prints data on host
    num_print = (num_print > size) ? size : num_print;
    std::cout << "host data: ";
    for(int i = 0; i < num_print; i++){
        std::cout << data[i] << ", ";
    }
    if(num_print < size)
        std::cout << "...";
    std::cout << "\n";
}

void print_device_data(int *device_data, int size, int num_print)
{
    // Prints data on device
    // print_device_data(device_data+N-num_print, num_print, num_print);
    if(num_print > size){
      // Print whole thing
      int* inarray = new int[size];
      hipMemcpy(inarray, device_data, size*sizeof(int), hipMemcpyDeviceToHost);
      std::cout << "(device) ";
      for(int i = 0; i < num_print; i++){
          std::cout << inarray[i] << ", ";
      }
      std::cout << "\n";
    }
    else
    {
      // Print first and last data
      int num_print1 = num_print/2;
      int num_print2 = num_print - num_print1;
      int* inarray1 = new int[num_print1];
      int* inarray2 = new int[num_print2];
      hipMemcpy(inarray1, device_data, num_print1*sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(inarray2, device_data+size-num_print2, num_print2*sizeof(int), hipMemcpyDeviceToHost);
      std::cout << "(device) ";
      for(int i = 0; i < num_print1; i++){
          std::cout << inarray1[i] << ", ";
      }
      std::cout << "...\n... ";
      for(int i = 0; i < num_print2; i++){
          std::cout << inarray2[i] << ", ";
      }
      std::cout << "\n";
    }
}

void exclusive_scan_iterative(int* data, int length)
{
    int N = length;
    // upsweep phase.
    std::cout << "Before upsweep: ";
    print_host_data(data, length, num_print);
    for (int twod = 1; twod < N; twod*=2)
    {
        int twod1 = twod*2;
        for(int i = 0; i < N; i += twod1)
            data[i+twod1-1] += data[i+twod-1];
    }
    std::cout << "After upsweep: ";
    print_host_data(data, length, num_print);
    data[N-1] = 0;
    // downsweep phase.
    for (int twod = N/2; twod >= 1; twod /= 2)
    {
        int twod1 = twod*2;
        for(int i = 0; i < N; i += twod1)
        {
            int t = data[i+twod-1];
            data[i+twod-1] = data[i+twod1-1];
            // change twod1 below to twod to reverse prefix sum.
            data[i+twod1-1] += t;
        }
    }
    std::cout << "After downsweep: ";
    print_host_data(data, length, num_print);
    std::cout << "\n";
}

__global__ void upsweep_kernel(int *device_data, int N, int twod)
{
    int twod1 = twod*2;
    long index = (blockIdx.x * blockDim.x + threadIdx.x) * (long)twod1;
    if ((index+twod1-1) < N)
        device_data[index+twod1-1] += device_data[index+twod-1];
}

__global__ void downsweep_kernel(int *device_data, int N, int twod)
{
    int twod1 = twod*2;
    long index = (blockIdx.x * blockDim.x + threadIdx.x) * (long)twod1;
    if((index+twod1-1) < N)
    {
        int t = device_data[index+twod-1];
        device_data[index+twod-1] = device_data[index+twod1-1];
        // change twod1 below to twod to reverse prefix sum.
        device_data[index+twod1-1] += t;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    const int N = nextPow2(length);
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    // {
    //   // DEBUGGING
    //   printf("threads = %d, blocks = %d, prod = %d\n", threadsPerBlock, blocks, threadsPerBlock*blocks);
    //   long max_index = (threadsPerBlock*blocks-1)*65536*2;
    //   std::cout << "Max index  = " << max_index << "\n";
    //   printf("Sizeof(long) = %ld\n", sizeof(long long));
    // }
    // upsweep phase.
    // {
    //     // DEBUGGING
    //     hipMemset(device_data+length, 0, (N-length)*sizeof(int));
    //     std::cout << "\tBefore upsweep: ";
    //     print_device_data(device_data, N, num_print); 
    // }
    for (int twod = 1; twod < N; twod*=2)
    {
        upsweep_kernel<<<blocks, threadsPerBlock>>>(device_data, N, twod);
        // {
        //     // DEBUGGING
        //     printf("\tDuring upsweep (twod = %d): ", twod);
        //     print_device_data(device_data, N, num_print);
        // }
    }
    // {
    //     // DEBUGGING
    //     std::cout << "\tAfter upsweep: ";
    //     print_device_data(device_data, N, num_print);
    // }
    // Zero unused memory
    hipMemset(device_data+length-1, 0, (N-length+1)*sizeof(int));
    // {
    //     // DEBUGGING
    //     std::cout << "\tAfter zeroing: ";
    //     print_device_data(device_data, N, num_print);
    // }

    // downsweep phase.
    for (int twod = N/2; twod >= 1; twod /= 2)
    {
        downsweep_kernel<<<blocks, threadsPerBlock>>>(device_data, N, twod);
        // {
        //     // DEBUGGING
        //     printf("\tDuring downsweep (twod = %d): ", twod);
        //     print_device_data(device_data, N, num_print);
        // }
    }
    // {
    //     // DEBUGGING
    //     std::cout << "\tAfter downsweep: ";
    //     print_device_data(device_data, N, num_print);
    //     std::cout << "\n";
    // }

}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    // {
    //     // DEBUGGING
    //     int length = end-inarray;
    //     int* test_array = new int[length];
    //     memcpy(test_array, inarray, length * sizeof(int));
    //     exclusive_scan_iterative(test_array, length);
    // }

    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}



int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    return 0;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
