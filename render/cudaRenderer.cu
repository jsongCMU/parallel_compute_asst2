#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#include <thrust/scan.h>
#include <thrust/device_ptr.h>

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
                hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants
{

    SceneName sceneName;

    int numberOfCircles;

    float *position;
    float *velocity;
    float *color;
    float *radius;

    int imageWidth;
    int imageHeight;
    float *imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// Read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// Color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// Include parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake()
{

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a)
{

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update positions of fireworks
__global__ void kernelAdvanceFireWorks()
{
    const float dt = 1.f / 60.f;
    const float pi = M_PI;
    const float maxDist = 0.25f;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;
    float *radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS)
    { // firework center; no update
        return;
    }

    // Determine the firework center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i + 1];

    // Update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j + 1] += velocity[index3j + 1] * dt;

    // Firework sparks
    float sx = position[index3j];
    float sy = position[index3j + 1];

    // Compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // Compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist)
    { // restore to starting position
        // Random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j + 1] = position[index3i + 1] + y;
        position[index3j + 2] = 0.0f;

        // Travel scaled unit length
        velocity[index3j] = cosA / 5.0;
        velocity[index3j + 1] = sinA / 5.0;
        velocity[index3j + 2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // Place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff)
    {
        radius[index] = 0.02f;
    }
    else
    {
        radius[index] += 0.01f;
    }
}

// kernelAdvanceBouncingBalls
//
// Update the position of the balls
__global__ void kernelAdvanceBouncingBalls()
{
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3 + 1];
    float oldPosition = position[index3 + 1];

    if (oldVelocity == 0.f && oldPosition == 0.f)
    { // stop-condition
        return;
    }

    if (position[index3 + 1] < 0 && oldVelocity < 0.f)
    { // bounce ball
        velocity[index3 + 1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3 + 1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3 + 1] += velocity[index3 + 1] * dt;

    if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon && oldPosition < 0.0f && fabsf(position[index3 + 1] - oldPosition) < epsilon)
    { // stop ball
        velocity[index3 + 1] = 0.f;
        position[index3 + 1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// Move the snowflake animation forward one time step.  Update circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake()
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float *positionPtr = &cuConstRendererParams.position[index3];
    float *velocityPtr = &cuConstRendererParams.velocity[index3];

    // Load from global memory
    float3 position = *((float3 *)positionPtr);
    float3 velocity = *((float3 *)velocityPtr);

    // Hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // Add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // Drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // Update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // Update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // If the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ((position.y + radius < 0.f) ||
        (position.x + radius) < -0.f ||
        (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // Restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // Store updated positions and velocities to global memory
    *((float3 *)positionPtr) = position;
    *((float3 *)velocityPtr) = velocity;
}


__device__ __inline__ void
shadePixel(float2 pixelCenter, float3 p, float4 *colorPtr, int circleIndex, int index3)
{

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    // Circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // There is a non-zero contribution.  Now compute the shading value

    // Suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks, etc., to implement the conditional.  It
    // would be wise to perform this logic outside of the loops in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME)
    {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f - p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);
    }
    else
    {
        // Simple: each circle has an assigned color
        rgb = *(float3 *)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    
    float oneMinusAlpha = 1.f - alpha;
    
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * colorPtr->x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * colorPtr->y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * colorPtr->z;
    newColor.w = alpha + colorPtr->w;

    *colorPtr = newColor;
}

__device__ __inline__ int
circleInBox(
    float circleX, float circleY, float circleRadius,
    float boxL, float boxR, float boxT, float boxB)
{

    // clamp circle center to box (finds the closest point on the box)
    float closestX = (circleX > boxL) ? ((circleX < boxR) ? circleX : boxR) : boxL;
    float closestY = (circleY > boxB) ? ((circleY < boxT) ? circleY : boxT) : boxB;

    // is circle radius less than the distance to the closest point on
    // the box?
    float distX = closestX - circleX;
    float distY = closestY - circleY;

    if (((distX * distX) + (distY * distY)) <= (circleRadius * circleRadius))
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void binCircles(int *imageBins, int binDim)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int numCircles = cuConstRendererParams.numberOfCircles;

    if (index >= numCircles)
        return;

    int index3 = 3 * index;

    // Read position and radius
    float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
    float rad = cuConstRendererParams.radius[index];

    float inv_bin = 1.0f / binDim;
    float x_padding = 0.5/cuConstRendererParams.imageWidth;
    float y_padding = 0.5/cuConstRendererParams.imageHeight;
    // For all bin regions, check if circle is inside
    for (int i = 0; i < binDim; i++)
    {
        float boxL = i * inv_bin;
        float boxR = (i + 1) * inv_bin + x_padding - 1e-12;

        if (i == binDim - 1)
            boxR = 1.0f;

        for (int j = 0; j < binDim; j++)
        {
            float boxB = j * inv_bin;
            float boxT = (j + 1) * inv_bin + y_padding - 1e-12;

            if (j == binDim - 1)
                boxT = 1.0f;

            if (circleInBox(p.x, p.y, rad, boxL, boxR, boxT, boxB))
                imageBins[i * numCircles + j * binDim * numCircles + index] = 1;
            else
                imageBins[i * numCircles + j * binDim * numCircles + index] = 0;
        }
    }
}

__global__ void getCircleIndexes(int *scanArr, int *result, int binDim)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int numCircles = cuConstRendererParams.numberOfCircles;

    if (index >= binDim * binDim * numCircles)
        return;

    int cur = scanArr[index];
    int circleIdx = index % numCircles;

    if (circleIdx == 0)
        return;

    int valueOffset = 0;
    
    int startOfBin = index - circleIdx;

    if (index >= numCircles)
        valueOffset = scanArr[index - circleIdx - 1];

    if (cur > scanArr[index - 1])
    {
        result[startOfBin + (cur - valueOffset) - 1] = circleIdx;
    }
}

__global__ void renderPixel(int *fullIndexArr, int *fullScanArr, int binDim)
{
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int imageWidth = cuConstRendererParams.imageWidth;
    int imageHeight = cuConstRendererParams.imageHeight;

    if (imageX >= imageWidth || imageY >= imageHeight)
        return;

    int numberOfCircles = cuConstRendererParams.numberOfCircles;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    int binX = imageX / ((float)imageWidth / binDim);
    int binY = imageY / ((float)imageHeight / binDim);
    int *indexArr = fullIndexArr + binX * numberOfCircles + binY * binDim * numberOfCircles;
    int *scanArr = fullScanArr + binX * numberOfCircles + binY * binDim * numberOfCircles;

    // The last element of our scan array holds the total number of circles
    int numCirclesInBin = scanArr[numberOfCircles - 1];

    // Need to subtract all previous bins total to get correct number
    if (!(binX == 0 && binY == 0))
    {
        numCirclesInBin -= (scanArr - 1)[0];
    }

    float4 color = *(float4 *)(&cuConstRendererParams.imageData[4 * (imageY * imageWidth + imageX)]);
    
    for (int i = 0; i < numCirclesInBin; i++)
    {
        int idx = indexArr[i];
        int index3 = idx * 3;

        float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
        float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(imageX) + 0.5f),
                                             invHeight * (static_cast<float>(imageY) + 0.5f));

        shadePixel(pixelCenterNorm, p, &color, idx, index3);
    }

    *(float4 *)(&cuConstRendererParams.imageData[4 * (imageY * imageWidth + imageX)]) = color;
}

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer()
{
    image = NULL;

    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;

    imageBins = NULL;
    scanArr = NULL;
    indexArr = NULL;
}

CudaRenderer::~CudaRenderer()
{

    if (image)
    {
        delete image;
    }

    if (position)
    {
        delete[] position;
        delete[] velocity;
        delete[] color;
        delete[] radius;
    }

    if (cudaDevicePosition)
    {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);

        hipFree(imageBins);
        hipFree(scanArr);
        hipFree(indexArr);
    }
}

const Image *
CudaRenderer::getImage()
{

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadScene(SceneName scene)
{
    sceneName = scene;
    loadCircleScene(sceneName, numberOfCircles, position, velocity, color, radius);
}

void CudaRenderer::setup()
{

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);

    if (numberOfCircles < 10)
        binDim = 4;
    else
        binDim = image->width / 50 + 2; 

    // Issues with image sizes (e.g. 256) can sometimes be fixed with even/odd image sizes
    // if (binDim % 2 == 1)
    //     binDim += 1;

    // Allocate memory to arrays
    cudaCheckError(hipMalloc(&imageBins, binDim * binDim * numberOfCircles * sizeof(int)));
    cudaCheckError(hipMalloc(&scanArr, binDim * binDim * numberOfCircles * sizeof(int)));
    cudaCheckError(hipMalloc(&indexArr, binDim * binDim * numberOfCircles * sizeof(int)));

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numberOfCircles = numberOfCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // Also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int *permX;
    int *permY;
    float *value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // Copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height)
{

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage()
{

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME)
    {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    }
    else
    {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES)
    {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    }
    else if (sceneName == BOUNCING_BALLS)
    {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    }
    else if (sceneName == HYPNOSIS)
    {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    }
    else if (sceneName == FIREWORKS)
    {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void CudaRenderer::render()
{
    // 256 threads per block is a healthy number
    int threadsPerBlock = 256;
    int blocks = (numberOfCircles + threadsPerBlock - 1) / threadsPerBlock;

    binCircles<<<blocks, threadsPerBlock>>>(imageBins, binDim);
    cudaCheckError(hipDeviceSynchronize());
    thrust::device_ptr<int> start = thrust::device_pointer_cast(imageBins);
    thrust::device_ptr<int> result = thrust::device_pointer_cast(scanArr);

    blocks = (binDim * binDim * numberOfCircles + threadsPerBlock - 1) / threadsPerBlock;
    thrust::inclusive_scan(start, start + binDim * binDim * numberOfCircles, result);
    cudaCheckError(hipDeviceSynchronize());
    getCircleIndexes<<<blocks, threadsPerBlock>>>(scanArr, indexArr, binDim);
    cudaCheckError(hipDeviceSynchronize());

    // int test[numberOfCircles];
    // hipMemcpy(test, indexArr + 1 * numberOfCircles, numberOfCircles * sizeof(int),
    //            hipMemcpyDeviceToHost);

    // for (int i = 0; i < numberOfCircles; i++)
    // {
    //     printf("%d |", test[i]);
    // }
    // printf("\n");

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);
    renderPixel<<<gridDim, blockDim>>>(indexArr, scanArr, binDim);
    cudaCheckError(hipDeviceSynchronize());
}
